#include "hip/hip_runtime.h"
//
// Created by goforbroke on 26.12.2020.
//

#include <cstdlib>

#include "../init.h"

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b;

    // Allocate memory
    a = (float *) malloc(sizeof(float) * N);
    b = (float *) malloc(sizeof(float) * N);
    out = (float *) malloc(sizeof(float) * N);

    hipMalloc((void **) &d_a, sizeof(float) * N);
    hipMalloc((void **) &d_b, sizeof(float) * N);

    // Initialize array
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    vector_add<<<256, 256>>>(out, d_a, d_b, N);

    hipFree(d_a);
    hipFree(d_b);
    free(a);
}