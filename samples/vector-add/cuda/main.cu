//
// Created by goforbroke on 26.12.2020.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {
    size_t index = threadIdx.x;
    size_t stride = blockDim.x;

    for (int load = index; load < n; load += stride) { // fake addition load
        for (int i = index; i < n; i += stride) {
            out[i] = a[i] + b[i];
        }
    }
}

int main(int argc, char **argv) {
    size_t NSAMPLES = atoi(argv[1]);

    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory
    a = (float *) malloc(sizeof(float) * NSAMPLES);
    b = (float *) malloc(sizeof(float) * NSAMPLES);
    out = (float *) malloc(sizeof(float) * NSAMPLES);

    hipMalloc((void **) &d_a, sizeof(float) * NSAMPLES);
    hipMalloc((void **) &d_b, sizeof(float) * NSAMPLES);
    hipMalloc((void **) &d_out, sizeof(float) * NSAMPLES);

    // Initialize array
    for (long i = 0; i < NSAMPLES; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMemcpy(d_a, a, sizeof(float) * NSAMPLES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * NSAMPLES, hipMemcpyHostToDevice);

    vector_add<<<256, 256>>>(d_out, d_a, d_b, NSAMPLES);

    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess)
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
    hipProfilerStop();

    hipMemcpy(out, d_out, sizeof(float) * NSAMPLES, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);
}