#include "hip/hip_runtime.h"
//
// Created by goforbroke on 26.12.2020.
//

#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

#include "../init.h"

__global__ void vector_add(float *out, float *a, float *b, int n) {
    size_t index = threadIdx.x;
    size_t stride = blockDim.x;

    for (int i = index; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory
    a = (float *) malloc(sizeof(float) * N);
    b = (float *) malloc(sizeof(float) * N);
    out = (float *) malloc(sizeof(float) * N);

    hipMalloc((void **) &d_a, sizeof(float) * N);
    hipMalloc((void **) &d_b, sizeof(float) * N);
    hipMalloc((void **) &d_out, sizeof(float) * N);

    // Initialize array
    for (long i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<N / 256, 256>>>(d_out, d_a, d_b, N);

    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess)
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
    hipProfilerStop();

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);
}