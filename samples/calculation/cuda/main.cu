#include "hip/hip_runtime.h"
//
// Created by goforbroke on 25.12.2020.
//

#include <cstdio>
#include "../init.h"

__global__ void initCalculation(float *buf, size_t n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    buf[i] = 1.0f * i / n;
    for (int j = 0; j < M; j++) { // fake addition load
        buf[i] = buf[i] * buf[i] - 0.25f;
    }
}

int main(int argc, char **argv) {
    size_t NSAMPLES = atoi(argv[1]);

    float data[NSAMPLES];
    float *d_data;
    hipMalloc(&d_data, NSAMPLES * sizeof(float));
    initCalculation<<<NSAMPLES / 256, 256>>>(d_data, NSAMPLES);
    hipMemcpy(data, d_data, NSAMPLES * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_data);

//    int sel;
//    printf("Enter an index: ");
//    scanf("%d", &sel);
//    printf("data[%d] = %f\n", sel, data[sel]);

    int sel = 100;
    printf("data[%d] = %f\n", sel, data[sel]);
}
