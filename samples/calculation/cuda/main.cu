#include "hip/hip_runtime.h"
//
// Created by goforbroke on 25.12.2020.
//

//#include <iostream>
//
//int main() {
//    std::cout << "Hello, World!" << std::endl;
//    return 0;
//}

#include <cstdio>

#include "../init.h"

__global__ void initCalculation(float *buf) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    buf[i] = 1.0f * i / N;
    for (int j = 0; j < M; j++)
        buf[i] = buf[i] * buf[i] - 0.25f;
}

int main() {
    float data[N];
    float *d_data;
    hipMalloc(&d_data, N * sizeof(float));
    initCalculation<<<N / 256, 256>>>(d_data);
    hipMemcpy(data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_data);

//    int sel;
//    printf("Enter an index: ");
//    scanf("%d", &sel);
//    printf("data[%d] = %f\n", sel, data[sel]);

    int sel = 100;
    printf("data[%d] = %f\n", sel, data[sel]);
}
